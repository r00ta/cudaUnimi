#include "hip/hip_runtime.h"
/* CUDA threads may access data from multiple memory spaces during their execution as illustrated by Figure 7.
				thread
					|								 ___________
					\								|Per Thread |
					|<------------------------------>|     local |
					/								|__memory___|
					\ 
					
				thread block
				\\\\\\\\\\\								_________________
				///////////<-------------------------->|                 |
				\\\\\\\\\\\<-------------------------->|Per block shared |
				///////////<-------------------------->|    memory       |
				\\\\\\\\\\\							   |_________________|
													_____________________________
				Grid 0 <------------------------->  |							 |
				Grid 1<-------------------------->  |Global memory 				 |
				....<---------------------------->  |							 |
				Grid n<-------------------------->  |____________________________|


				Thread Block
Each thread has private local memory. Each thread block has shared memory visible to all threads of the block 
and with the same lifetime as the block. All threads have access to the same global memory.

There are also two additional read-only memory spaces accessible by all threads: the constant and texture memory spaces.
 The global, constant, and texture memory spaces are optimized for different memory usages (see Device Memory Accesses). 
 Texture memory also offers different addressing modes, as well as data filtering, for some specific data formats 
 (see Texture and Surface Memory).

The global, constant, and texture memory spaces are persistent across kernel launches by the same application.

 As illustrated by Figure 8, the CUDA programming model assumes that the CUDA threads execute on a physically separate device that operates 
 as a coprocessor to the host running the C program. This is the case, for example, when the kernels execute on a GPU and the rest of the C 
 program executes on a CPU.

The CUDA programming model also assumes that both the host and the device maintain their own separate memory spaces in DRAM, referred to as 
host memory and device memory, respectively. Therefore, a program manages the global, constant, and texture memory spaces visible to kernels 
through calls to the CUDA runtime (described in Programming Interface). This includes device memory allocation and deallocation as well as data 
transfer between host and device memory.

The compute capability of a device is represented by a version number, also sometimes called its "SM version". 
This version number identifies the features supported by the GPU hardware and is used by applications at runtime to determine which hardware features and/or instructions are available on the present GPU.

The compute capability comprises a major revision number X and a minor revision number Y and is denoted by X.Y.

Devices with the same major revision number are of the same core architecture. The major revision number is 5 for devices based on the
 Maxwell architecture, 3 for devices based on the Kepler architecture, 2 for devices based on the Fermi architecture, and 1 for devices
  based on the Tesla architecture.

The minor revision number corresponds to an incremental improvement to the core architecture, possibly including new features.

As mentioned in Heterogeneous Programming, the CUDA programming model assumes a system composed of a host and a device, 
each with their own separate memory. Kernels operate out of device memory, so the runtime provides functions to allocate,
deallocate, and copy device memory, as well as transfer data between host memory and device memory.

Device memory can be allocated either as linear memory or as CUDA arrays.

CUDA arrays are opaque memory layouts optimized for texture fetching. They are described in Texture and Surface Memory.

Linear memory exists on the device in a 40-bit address space, so separately allocated entities can reference one another via pointers, 
for example, in a binary tree.

Linear memory is typically allocated using hipMalloc() and freed using hipFree() and data transfer between host memory and device memory 
are typically done using hipMemcpy(). In the vector addition code sample of Kernels, the vectors need to be copied from host memory to
device memory:
*/

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
            
// Host code
int main()
{
    int N = ...;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);

    // Initialize input vectors
    ...

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
            
    // Free host memory
    ...
}

/*
Linear memory can also be allocated through hipMallocPitch() and hipMalloc3D(). These functions are recommended for allocations of 2D or 3D arrays as it makes sure that the allocation is appropriately padded to meet the alignment requirements described in Device Memory Accesses, therefore ensuring best performance when accessing the row addresses or performing copies between 2D arrays and other regions of device memory (using the hipMemcpy2D() and hipMemcpy3D() functions). The returned pitch (or stride) must be used to access array elements. The following code sample allocates a width x height 2D array of floating-point values and shows how to loop over the array elements in device code:

// Host code
int width = 64, height = 64;
float* devPtr;
size_t pitch;
hipMallocPitch(&devPtr, &pitch,
                width * sizeof(float), height);
MyKernel<<<100, 512>>>(devPtr, pitch, width, height);

// Device code
__global__ void MyKernel(float* devPtr,
                         size_t pitch, int width, int height)
{
    for (int r = 0; r < height; ++r) {
        float* row = (float*)((char*)devPtr + r * pitch);
        for (int c = 0; c < width; ++c) {
            float element = row[c];
        }
    }
}

The following code sample allocates a width x height x depth 3D array of floating-point values and shows how to loop over the array elements in device code:

// Host code
int width = 64, height = 64, depth = 64;
hipExtent extent = make_hipExtent(width * sizeof(float),
                                    height, depth);
hipPitchedPtr devPitchedPtr;
hipMalloc3D(&devPitchedPtr, extent);
MyKernel<<<100, 512>>>(devPitchedPtr, width, height, depth);

// Device code
__global__ void MyKernel(hipPitchedPtr devPitchedPtr,
                         int width, int height, int depth)
{
    char* devPtr = devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * height;
    for (int z = 0; z < depth; ++z) {
        char* slice = devPtr + z * slicePitch;
        for (int y = 0; y < height; ++y) {
            float* row = (float*)(slice + y * pitch);
            for (int x = 0; x < width; ++x) {
                float element = row[x];
            }
        }
    }
}

The reference manual lists all the various functions used to copy memory between linear memory allocated with hipMalloc(), linear memory allocated with hipMallocPitch() or hipMalloc3D(), CUDA arrays, and memory allocated for variables declared in global or constant memory space.

The following code sample illustrates various ways of accessing global variables via the runtime API:

__constant__ float constData[256];
float data[256];
hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));

__device__ float devData;
float value = 3.14f;
hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));

__device__ float* devPointer;
float* ptr;
hipMalloc(&ptr, 256 * sizeof(float));
hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));

hipGetSymbolAddress() is used to retrieve the address pointing to the memory allocated for a variable declared in global memory space. The size of the allocated memory is obtained through hipGetSymbolSize()
*/