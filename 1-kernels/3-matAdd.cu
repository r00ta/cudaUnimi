/*The number of threads per block and the number of blocks per grid specified in the <<<...>>> syntax can be of type int or dim3.
 Two-dimensional blocks or grids can be specified as in the example above.

Each block within the grid can be identified by a one-dimensional, two-dimensional, or three-dimensional index accessible 
within the kernel through the built-in blockIdx variable. The dimension of the thread block is accessible within the kernel
through the built-in blockDim variable.

Extending the previous MatAdd() example to handle multiple blocks, the code becomes as follows.

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1024
__device__ int A[N][N];
__device__ int B[N][N];
__device__ int C[N][N];

__global__ void MatAdd()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N)
        C[i][j] = A[i][j] + B[i][j];
}

int main()
{
    // Kernel invocation
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    MatAdd<<<numBlocks, threadsPerBlock>>>();
        hipDeviceSynchronize();
}

/*A thread block size of 16x16 (256 threads), although arbitrary in this case, is a common choice. The grid is created with enough 
blocks to have one thread per matrix element as before. For simplicity, this example assumes that the number of threads per 
grid in each dimension is evenly divisible by the number of threads per block in that dimension, although that need not be 
the case.

Thread blocks are required to execute independently: It must be possible to execute them in any order, in parallel or in series. 
This independence requirement allows thread blocks to be scheduled in any order across any number of cores as illustrated by 
Figure 5, enabling programmers to write code that scales with the number of cores.

Threads within a block can cooperate by sharing data through some shared memory and by synchronizing their execution to
 coordinate memory accesses. More precisely, one can specify synchronization points in the kernel by calling the __syncthreads() 
 intrinsic function; __syncthreads() acts as a barrier at which all threads in the block must wait before any is allowed to
  proceed. Shared Memory gives an example of using shared memory.


For efficient cooperation, the shared memory is expected to be a low-latency memory near each processor core 
(much like an L1 cache) and __syncthreads() is expected to be lightweight.
*/